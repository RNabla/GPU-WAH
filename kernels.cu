#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include ""
#include "kernels.h"
#include "tests.h"
#include <stdio.h>
#include "const.h"

// some defines to avoid repetition
#define IF_LAST if(id == (warpSize - 1))

#define WARP_OPERATION_DOWN(OP, NAME) \
__inline__ __device__ int NAME(int val){ \
	for (int mask = WARP_SIZE/2; mask > 0; mask /= 2)\
		val OP __shfl_xor(val, mask);\
	return val; \
}\

WARP_OPERATION_DOWN(|=, orWithinWarp);

__inline__ __device__ int localScan(int val, int id){
	for(int i = 1; i < 32; i<<=1){
		int ret = __shfl_up(val, i);
		val += id >= i ? ret : 0;
	}
	return val;
}

__inline__ __device__ void markWordTypes(int w, int* end, int* beg, int id){
	IF_LAST{
		end[threadIdx.y] = w;
	}
	IF_LAST{
		beg[threadIdx.y] = w;
	}
}


__inline__ __device__ void writeEndingSize(int id, int* lengths, int size){
	IF_LAST{
		lengths[threadIdx.y] = size;
	}
}

__global__ void compressData(unsigned int* data, unsigned int* output) {
	// count of words for every warp
	__shared__ int counts[32];
	// lenght of the last word in a warp
	__shared__ int endLengths[32];
	// type of the last word in a warp
	__shared__ int endings[32];
	// type of the first word in a warp
	__shared__ int beginnings[32];
	// array indicating whether the last thread of a warp has been merged
	__shared__ bool merged[32] = {false};
	// shift related to warpmerging
//	__shared__ int mergeShifts[32] = {0};


	// get thread id
	int id = threadIdx.x;
	int id_global = threadIdx.y *31 + id;
	unsigned int word = 0;
	// retrieve word, only first 31 threads
	if (id < WARP_SIZE - 1) {
		word = data[id_global];
	}
	// divide words into 31bit parts 
	// gets 31 - id bits from one lane above
	// and id bits from own lane
	//word = (__shfl_down(word, 1) & (ONES31 >> id)) << id | (word & TOP31ONES) >> (32 - id);
	word = ONES31 & ((__shfl_up(word, 1) >> (32 - id)) | (word << id));


	// word info variables
	int ones = 0;
	int zeros = 0;
	int literals = 0;

	// mark word types for warp
	// detect words with zeros and words with ones
	// is a zero fill word
	if (word == ZEROS) {
		zeros |= 1 << id;
		markWordTypes(WORD_ZEROS, endings, beginnings, id);
	}

	// is a one fill word
	else if (word == ONES31) {
		ones |= 1 << id;
		markWordTypes(WORD_ONES, endings, beginnings, id);
	}
	else
	{
		markWordTypes(WORD_LITERAL, endings, beginnings, id);
	}

	// exchange word information within the warp
	zeros = orWithinWarp(zeros);
	ones = orWithinWarp(ones);
	literals = ~(zeros | ones);

	// send complete information to other threads
	if (id == WARP_LEADER) {
		zeros == __shfl(zeros, 0);
		ones == __shfl(ones, 0);
		literals == __shfl(literals, 0);
	}

	int n = 0x3 << id;
	int flags = BIT31;
	bool idle = true;
	// if is not last
	if (id < 31) {
		int res = 1 << id;
		if (((n & zeros) == res || (n & ones) == res || (literals & (1 << id)) > 0)) {
			// mark endings
			flags |= 1 << id;
			idle = false;
		}
	}
	else{
		idle = false;
	}
	// exchange endings 
	flags = orWithinWarp(flags);
	int blockSize = 1;

	// calculate the number of words within a block
	if (!idle) {
		for (int i = id-1; i > 0; i--) {
			if ((flags & (1 << i)) > 0) {
				break;
			}
			blockSize++;
		}
		if (word == ONES31) {
//			word = BIT3130 | blockSize;
			writeEndingSize(id, endLengths, blockSize);
		}
		else if (word == ZEROS) {
//			word = BIT31 | blockSize;
			writeEndingSize(id, endLengths, blockSize);
		}
	}

	// last thread calculates the number of words and writes it to the shared array
	if(id == WARP_LEADER){
		counts[threadIdx.y] = __popc(flags);
	}

	// sync all threads within block
	__syncthreads();

	// the first warp scans the array and gets total block word size
	// then calculates offset
	int mergeShift = 0;
	if(threadIdx.y == BLOCK_LEADER){
		int count = counts[id];
		// only execute if it's a non
		if((threadIdx.y == warpSize - 1) || (endings[id] != beginnings[id+1])){
			int i = 0;
			// calculate merge shifts
			while(true){
				// has 1 length and words match
				if(i < id && counts[id - i] == 1 && beginnings[id] == endings[id-i]){
					mergeShift++;
					merged[id - i] = true;

				}
				else if(i <= id && beginnings[id] == endings[id - i]){
					mergeShift++;
					merged[id - i] = true;
				}
				else break;
			}
			mergeShift = localScan(mergeShift, id);
			int globalOffset = localScan(count, id);
			counts[id] = globalOffset - count - mergeShift;
		}
	}

	__syncthreads();
	// get global offset for warp and warp offset
	int index = counts[threadIdx.y] + __popc(((1<<id) - 1) & flags);
	output[index] = word;


}




