#include "decompress.h"
#include "kernels.h"
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include "timeMeasuring.h"

unsigned int* decompress(
		unsigned int* data,
		unsigned long long int dataSize,
		unsigned long long int* outSize,
		unsigned long long int* offsets,
		unsigned long long int* blockSizes,
		unsigned int blocks,
		float* pTransferToDeviceTime,
		float* pCompressionTime,
		float* ptranserFromDeviceTime){

	// times to be measured
	float transferToDeviceTime;
	float compressionTime;
	float transferFromDeviceTime;

	// start measuring time
	CREATE_TIMER
	START_TIMER

	unsigned int *data_gpu, *result_gpu, *finalOutput_gpu, *output_cpu;
	unsigned long long int *offsets_gpu, *blockSizes_gpu;
	unsigned long long int blockCount = dataSize / 1024;

	if(dataSize % 1024 > 0){
		blockCount++;
	}
	if(hipSuccess != hipMalloc((void**)&data_gpu, sizeof(int)*dataSize)){
		std::cout << "Decomp: Could not allocate space for data array" << std::endl;
	}
	if(hipSuccess != hipMalloc((void**)&offsets_gpu, sizeof(unsigned long long int)* blocks)){
		std::cout << "Decomp: Could not allocate space for offset array" << std::endl;
	}
	if(hipSuccess != hipMalloc((void**)&blockSizes_gpu, sizeof(unsigned long long int)* blocks)){
		std::cout << "Decomp: Could not allocate space for block sizes array" << std::endl;
	}
	hipMemcpy(data_gpu, data, sizeof(int)*dataSize, hipMemcpyHostToDevice);
	hipMemcpy(offsets_gpu, offsets, sizeof(unsigned long long int)*blocks, hipMemcpyHostToDevice);
	hipMemcpy(blockSizes_gpu, blockSizes, sizeof(unsigned long long int) * blocks, hipMemcpyHostToDevice);

	STOP_TIMER
	GET_RESULT(transferToDeviceTime)
	START_TIMER
//	counts_cpu = (unsigned int*) malloc(sizeof(int)*dataSize);
	dim3 blockDim(32, 32);
	// get blocked sizes
//	getCounts<<<blockCount,blockDim>>>(data_gpu, counts_gpu, dataSize);
//	unsigned long long int lastBlockSize;
//	hipMemcpy(&lastBlockSize, counts_gpu  + (dataSize - 1), sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	// scan block sizes
//	thrust::device_ptr<unsigned long long int> countsPtr(counts_gpu);
	// get counts
//	thrust::exclusive_scan(countsPtr, countsPtr + dataSize, countsPtr);
//	unsigned long long int lastOffset;
//	thrust::inclusive_scan(counts_cpu, counts_cpu + dataSize, counts_cpu);
//	hipMemcpy(&lastOffset, counts_gpu + (dataSize - 1), sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	unsigned long long int outputSize = 1024*blocks;
	unsigned long long int realSize = 31*outputSize;

	if(realSize % 32 > 0){
		realSize /=32;
		realSize++;
	}
	else{
		realSize /=32;
	}
	SAFE_ASSIGN(outSize, realSize);
//	free(counts_cpu);
	if(hipSuccess != hipMalloc((void**)&result_gpu, sizeof(int) * outputSize)){
		std::cout << "Decomp: Could not allocate space for results array" << std::endl;
	}

	decompressWords<<<blocks,blockDim>>>(data_gpu, result_gpu, offsets_gpu, blockSizes_gpu, blocks, dataSize);
	hipFree(data_gpu);
	hipFree(offsets_gpu);
	hipFree(blockSizes_gpu);

	blockCount = outputSize / 1024;

	if(dataSize % 1024 > 0){
		blockCount++;
	}

	hipError_t res = hipMalloc((void**)&finalOutput_gpu, sizeof(int)*outputSize);
	if(hipSuccess != res){
		std::cout << "Error" << std::endl;
		std::cout << hipGetErrorName(res) << std::endl;
		std::cout << "Decomp: Could not allocate space for final output array" << std::endl;

	}
	mergeWords<<<blockCount,blockDim>>>(result_gpu, finalOutput_gpu, outputSize);
	hipFree(result_gpu);

	STOP_TIMER
	GET_RESULT(compressionTime)
	START_TIMER

	output_cpu = (unsigned int*)malloc(sizeof(int) * outputSize);
	hipMemcpy(output_cpu, finalOutput_gpu, sizeof(int) * outputSize, hipMemcpyDeviceToHost);
	hipFree(finalOutput_gpu);
	STOP_TIMER
	GET_RESULT(transferFromDeviceTime)

	SAFE_ASSIGN(pCompressionTime, compressionTime);
	SAFE_ASSIGN(pTransferToDeviceTime, transferToDeviceTime);
	SAFE_ASSIGN(ptranserFromDeviceTime, transferFromDeviceTime);

	return output_cpu;
}

