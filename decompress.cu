#include "decompress.h"
#include "kernels.h"
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include "timeMeasuring.h"

unsigned int* decompress(
		unsigned int* data,
		unsigned long long int dataSize,
		unsigned long long int* outSize,
		unsigned long long int* offsets,
		unsigned long long int* blockSizes,
		unsigned int blocks,
		float* pTransferToDeviceTime,
		float* pCompressionTime,
		float* ptranserFromDeviceTime){

	// times to be measured
	float transferToDeviceTime;
	float compressionTime;
	float transferFromDeviceTime;

	// start measuring time
	CREATE_TIMER
	START_TIMER

	unsigned int *data_gpu, *result_gpu, *finalOutput_gpu, *output_cpu;
	unsigned long long int *offsets_gpu, *blockSizes_gpu;
	unsigned long long int blockCount = dataSize / 1024;

	if(dataSize % 1024 > 0){
		blockCount++;
	}
	hipMalloc((void**)&data_gpu, sizeof(int)*dataSize);
	hipMalloc((void**)&offsets_gpu, sizeof(unsigned long long int)* blocks);
	hipMalloc((void**)&blockSizes_gpu, sizeof(unsigned long long int)* blocks);
	hipMemcpy(data_gpu, data, sizeof(int)*dataSize, hipMemcpyHostToDevice);
	hipMemcpy(offsets_gpu, offsets, sizeof(unsigned long long int)*blocks, hipMemcpyHostToDevice);
	hipMemcpy(blockSizes_gpu, blockSizes, sizeof(unsigned long long int) * blocks, hipMemcpyHostToDevice);

	STOP_TIMER
	GET_RESULT(transferToDeviceTime)
	START_TIMER
//	counts_cpu = (unsigned int*) malloc(sizeof(int)*dataSize);
	dim3 blockDim(32, 32);
	// get blocked sizes
//	getCounts<<<blockCount,blockDim>>>(data_gpu, counts_gpu, dataSize);
//	unsigned long long int lastBlockSize;
//	hipMemcpy(&lastBlockSize, counts_gpu  + (dataSize - 1), sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	// scan block sizes
//	thrust::device_ptr<unsigned long long int> countsPtr(counts_gpu);
	// get counts
//	thrust::exclusive_scan(countsPtr, countsPtr + dataSize, countsPtr);
//	unsigned long long int lastOffset;
//	thrust::inclusive_scan(counts_cpu, counts_cpu + dataSize, counts_cpu);
//	hipMemcpy(&lastOffset, counts_gpu + (dataSize - 1), sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	unsigned long long int outputSize = 1024*blocks;
	unsigned long long int realSize = 31*outputSize;

	if(realSize % 32 > 0){
		realSize /=32;
		realSize++;
	}
	else{
		realSize /=32;
	}
	SAFE_ASSIGN(outSize, realSize);
//	free(counts_cpu);
	hipMalloc((void**)&result_gpu, sizeof(int) * outputSize);

	decompressWords<<<blocks,blockDim>>>(data_gpu, result_gpu, offsets_gpu, blockSizes_gpu, blocks, dataSize);
	hipFree(data_gpu);
	hipFree(offsets_gpu);
	hipFree(blockSizes_gpu);

	blockCount = outputSize / 1024;
	if(dataSize % 1024 > 0){
		blockCount++;
	}

	hipMalloc((void**)&finalOutput_gpu, sizeof(int)*outputSize);
	mergeWords<<<blockCount,blockDim>>>(result_gpu, finalOutput_gpu, outputSize);
	hipFree(result_gpu);

	STOP_TIMER
	GET_RESULT(compressionTime)
	START_TIMER

	output_cpu = (unsigned int*)malloc(sizeof(int) * outputSize);
	hipMemcpy(output_cpu, finalOutput_gpu, sizeof(int) * outputSize, hipMemcpyDeviceToHost);
	hipFree(finalOutput_gpu);
	STOP_TIMER
	GET_RESULT(transferFromDeviceTime)

	SAFE_ASSIGN(pCompressionTime, compressionTime);
	SAFE_ASSIGN(pTransferToDeviceTime, transferToDeviceTime);
	SAFE_ASSIGN(ptranserFromDeviceTime, transferFromDeviceTime);

	return output_cpu;
}

unsigned int* reorder(
		unsigned long long int* blockSizes,
		unsigned long long int* offsets,
		unsigned long long int blockCount,
		unsigned int* data,
		unsigned long long int dataSize,
		float* pTransferToDeviceTime,
		float* pReoderingTime,
		float* ptranserFromDeviceTime
		){

	// times to be measured
	float transferToDeviceTime;
	float reorderingTime;
	float transferFromDeviceTime;

	CREATE_TIMER
	START_TIMER

	unsigned int *data_gpu, *output_gpu;
	unsigned long long int* blockSizes_gpu, *offsets_gpu, *outputOffsets_gpu;
	// allocate gpu memory
	hipMalloc((void**)&blockSizes_gpu, sizeof(unsigned long long int)*blockCount);
	hipMalloc((void**)&offsets_gpu, sizeof(unsigned long long int)*blockCount);
	hipMalloc((void**)&data_gpu, sizeof(int)*dataSize);
	hipMalloc((void**)&output_gpu, sizeof(int)*dataSize);
	hipMalloc((void**)&outputOffsets_gpu, sizeof(unsigned long long int)*blockCount);

	hipMemcpy(blockSizes_gpu, blockSizes, sizeof(unsigned long long int) * blockCount, hipMemcpyHostToDevice);
	hipMemcpy(offsets_gpu, offsets, sizeof(unsigned long long int) * blockCount, hipMemcpyHostToDevice);
	hipMemcpy(data_gpu, data, sizeof(int) * dataSize, hipMemcpyHostToDevice);

	STOP_TIMER
	GET_RESULT(transferToDeviceTime)

	START_TIMER

	thrust::device_ptr<unsigned long long int> pBlockSizes(blockSizes_gpu);
	thrust::device_ptr<unsigned long long int> pOutputOffets(outputOffsets_gpu);

	thrust::exclusive_scan(pBlockSizes, pBlockSizes + blockCount, pOutputOffets);

	int b = dataSize / 1024;

	if(dataSize % 1024 > 0){
		b++;
	}

//	reoderKernel<<<b,<te dim3(32, 32)>>>(blockSizes_gpu, offsets_gpu, outputOffsets_gpu, blockCount, data_gpu, dataSize, output_gpu);

	STOP_TIMER
	GET_RESULT(reorderingTime)

	START_TIMER

	unsigned int* output = (unsigned int*) malloc(sizeof(int) * dataSize);
	hipMemcpy(output, output_gpu, sizeof(int) * dataSize, hipMemcpyDeviceToHost);
	hipFree(blockSizes_gpu);
	hipFree(offsets_gpu);
	hipFree(data_gpu);
	hipFree(outputOffsets_gpu);
	hipFree(output_gpu);


	STOP_TIMER
	GET_RESULT(transferFromDeviceTime)

	SAFE_ASSIGN(pReoderingTime, reorderingTime);
	SAFE_ASSIGN(pTransferToDeviceTime, transferToDeviceTime);
	SAFE_ASSIGN(ptranserFromDeviceTime, transferFromDeviceTime);

	return output;

}
