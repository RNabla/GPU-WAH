#include "hip/hip_runtime.h"
/*
 * NoSortPackage.cpp
 *
 *  Created on: Feb 13, 2019
 *      Author: holgus103
 */

#include "NoSortPackage.h"
#include "compress.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include "timeMeasuring.h"

template<class T>
NoSortPackage<T>::NoSortPackage() {

}

template<class T>
NoSortPackage<T>::~NoSortPackage() {

}


template<class T>
void NoSortPackage<T>::c_allocateMemory(){
	CompressedPackage<T>::c_allocateMemory();
	// allocate memory on the device
	if(hipSuccess != hipMalloc((void**)&(this->sizeCounter_gpu), sizeof(T))){
		std::cout << "Could not allocate space for size counter" << std::endl;
		this->c_cleanup();
		return;
	}
	if(hipSuccess != hipMalloc((void**)&(this->orderingArray_gpu), this->blockCount * sizeof(T))){
		std::cout << "Could not allocate space for order array" << std::endl;
		this->c_cleanup();
		return;
	}

}


template<class T>
void NoSortPackage<T>::c_runAlgorithm(){
		dim3 blockSize = dim3(32, 32, 1);
		// call compression kernel
		//compressData<<<this->blockCount,blockSize>>>(this->data_gpu, this->compressed_gpu, this->blockCounts_gpu, this->orderArray_gpu, this->sizeCounter_gpu, this->size);

		// remove unnecessary data
		hipFree((void*)this->data_gpu);

		// allocate memory for block sizes
		this->blockSizes = (T*)malloc(sizeof(T) *this->blockCount);

		// copy block sizes
		if(hipSuccess != hipMemcpy(this->blockSizes, this->blockCounts_gpu, this->blockCount * sizeof(T), hipMemcpyDeviceToHost)){
			std::cout << "Could not copy last block counts" << std::endl;
			this->c_cleanup();
			return;
		}

		// allocate ordering array
		this->orderingArray = (T*) malloc(sizeof(T) * this->blockCount);

		// copy ordering array
		if(hipSuccess != hipMemcpy(this->orderingArray, this->orderingArray_gpu, this->blockCount * sizeof(T), hipMemcpyDeviceToHost)){
			std::cout << "Could not copy ordering array" << std::endl;
			this->c_cleanup();
			return;
		}

		unsigned long long int outputSize = 0;;

		if(hipSuccess != hipMemcpy(&outputSize, this->sizeCounter_gpu, sizeof(T), hipMemcpyDeviceToHost)){
			std::cout << "Could not copy last block offset" << std::endl;
			this->c_cleanup();
			return;
		}
		this->compressedSize = outputSize;
}


template<class T>
void NoSortPackage<T>::c_cleanup(){
	if(this->data_gpu) hipFree(this->data_gpu);
	if(this->compressed_gpu) hipFree(this->compressed_gpu);
	if(this->blockCounts_gpu) hipFree(this->blockCounts_gpu);
	if(this->orderingArray_gpu) hipFree(this->orderingArray_gpu);
	if(this->sizeCounter_gpu) hipFree(this->sizeCounter_gpu);
}


template<class T>
void NoSortPackage<T>::d_runAlgorithm(){
	dim3 blockDim(32, 32);
	// get blocked sizes
	//getCounts<<<blockCount,blockDim>>>(this->data_gpu, this->blockCounts_gpu, this->compressedSize);
	unsigned long long int lastBlockSize;
	hipMemcpy(&lastBlockSize, this->blockCounts_gpu  + (this->compressedSize - 1), sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	// scan block sizes
	thrust::device_ptr<T> countsPtr(this->blockCounts_gpu);
	// get counts
	thrust::exclusive_scan(countsPtr, countsPtr + this->compressedSize, countsPtr);
	unsigned long long int lastOffset;
//	thrust::inclusive_scan(counts_cpu, counts_cpu + dataSize, counts_cpu);
	hipMemcpy(&lastOffset, this->blockCounts_gpu + (this->compressedSize - 1), sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	unsigned long long int outputSize = lastBlockSize + lastOffset;
	unsigned long long int realSize = 31*outputSize;

	if(realSize % 32 > 0){
		realSize /=32;
		realSize++;
	}
	else{
		realSize /=32;
	}
	this->decompressedSize = realSize;
//	free(counts_cpu);
	hipMalloc((void**)&(this->result_gpu), sizeof(int) * this->decompressedSize);

	//decompressWords<<<blockCount,blockDim>>>(this->data_gpu, this->blockCounts_gpu, this->result_gpu, this->compressedSize);
	hipFree(this->data_gpu);
	hipFree(this->counts_gpu);

	this->blockCount = outputSize / 1024;
	if(this->compressedSize % 1024 > 0){
		this->blockCount++;
	}

	hipMalloc((void**)&this->finalOutput_gpu, sizeof(int)*this->decompressedSize);
	//mergeWords<<<blockCount,blockDim>>>(this->result_gpu, this->finalOutput_gpu, this->decompressedSize);
	hipFree(this->result_gpu);
}


template<class T>
T* NoSortPackage<T>::getOrderingArray(){
	return this->orderingArray;
}

template<class T>
T NoSortPackage<T>::getOrderingLength(){
	return this->blockCount;
}

template<class T>
T* NoSortPackage<T>::getBlockSizes(){
	return this->blockSizes;
}


// void  NoSortPackage::compressData(unsigned int* data, unsigned long long int size){
	// // times to be measured
	// 	float transferToDeviceTime;
	// 	float compressionTime;
	// 	float transferFromDeviceTime;

	// 	// start measuring time
	// 	hipEvent_t start, stop;
	// 	hipEventCreate(&start);
	// 	hipEventRecord(start,0);

	// 	unsigned long long blockCount = dataSize / (31*32);

	// 	if(dataSize % (31*32)> 0){
	// 		blockCount++;
	// 	}
	// 	// assign blockCount
	// 	this->orderingLength = blockCount;

	// 	unsigned int *data_gpu, *compressed_gpu;
	// 	unsigned long long int *blockCounts_gpu, *sizeCounter_gpu, *orderArray_gpu;

	// 	// calculate max output size (one extra bit for every 31 bits)
	// 	unsigned long long int maxExpectedSize = 8*sizeof(int)*dataSize;
	// 	if(maxExpectedSize % 31 > 0){
	// 		maxExpectedSize /= 31;
	// 		maxExpectedSize++;
	// 	}
	// 	else{
	// 		maxExpectedSize /= 31;
	// 	}

	// 	dim3 blockSize = dim3(32, 32, 1);

	// 	// allocate memory on the device
	// 	if(hipSuccess != hipMalloc((void**)&sizeCounter_gpu, sizeof(unsigned long long int))){
	// 		std::cout << "Could not allocate space for size counter" << std::endl;
	// 		FREE_ALL
	// 		return NULL;
	// 	}
	// 	if(hipSuccess != hipMalloc((void**)&orderArray_gpu, blockCount * sizeof(unsigned long long int))){
	// 		std::cout << "Could not allocate space for order array" << std::endl;
	// 		FREE_ALL
	// 		return NULL;
	// 	}
	// 	if(hipSuccess != hipMalloc((void**)&data_gpu, dataSize * sizeof(int))){
	// 		std::cout << "Could not allocate space for the data" << std::endl;
	// 		FREE_ALL
	// 		return NULL;
	// 	}
	// 	if(hipSuccess != hipMalloc((void**)&compressed_gpu, maxExpectedSize * sizeof(int))){
	// 		std::cout << "Could not allocate space for the compressed output" << std::endl;
	// 		FREE_ALL
	// 		return NULL;
	// 	}
	// 	if(hipSuccess != hipMalloc((void**)&blockCounts_gpu, blockCount* sizeof(unsigned long long int))){
	// 		std::cout << "Could not allocate space for the block sizes" << std::endl;
	// 		FREE_ALL
	// 		return NULL;
	// 	}

	// 	// copy input
	// 	if(hipSuccess != hipMemcpy(data_gpu, data_cpu, dataSize*sizeof(int), hipMemcpyHostToDevice)){
	// 		std::cout << "Could not copy input" << std::endl;
	// 		FREE_ALL
	// 		return NULL;
	// 	}

	// 	// get transfer time
	// 	hipEventCreate(&stop);
	// 	hipEventRecord(stop,0);
	// 	hipEventSynchronize(stop);
		// hipEventElapsedTime(&transferToDeviceTime, start,stop);

		// restart time measuring
		// hipEventCreate(&start);
		// hipEventRecord(start,0);

		// // call compression kernel
		// compressData<<<blockCount,blockSize>>>(data_gpu, compressed_gpu, blockCounts_gpu, orderArray_gpu, sizeCounter_gpu, dataSize);

		// // remove unnecessary data
		// hipFree((void*)data_gpu);

		// // allocate memory for block sizes
		// this->blockSizes = (unsigned long long int*)malloc(sizeof(unsigned long long int) *blockCount);

		// // copy block sizes
		// if(hipSuccess != hipMemcpy(this->blockSizes, blockCounts_gpu, blockCount * sizeof(unsigned long long int), hipMemcpyDeviceToHost)){
		// 	std::cout << "Could not copy last block counts" << std::endl;
		// 	FREE_ALL
		// 	return NULL;
		// }

		// // allocate ordering array
		// this->orderingArray = (unsigned long long int*) malloc(sizeof(unsigned long long int) * blockCount);

		// // copy ordering array
		// if(hipSuccess != hipMemcpy(this->orderingArray, orderArray_gpu, blockCount * sizeof(unsigned long long int), hipMemcpyDeviceToHost)){
		// 	std::cout << "Could not copy ordering array" << std::endl;
		// 	FREE_ALL
		// 	return NULL;
		// }

		// unsigned long long int outputSize = 0;;

		// if(hipSuccess != hipMemcpy(&outputSize, sizeCounter_gpu, sizeof(unsigned long long int), hipMemcpyDeviceToHost)){
		// 	std::cout << "Could not copy last block offset" << std::endl;
		// 	FREE_ALL
		// 	return NULL;
		// }
		// this->compressedSize = outputSize;


		// get compression time
		// hipEventCreate(&stop);
		// hipEventRecord(stop,0);
		// hipEventSynchronize(stop);
		// hipEventElapsedTime(&compressionTime, start,stop);

		// // restart time measuring
		// hipEventCreate(&start);
		// hipEventRecord(start,0);

		// // allocate memory for results
		// unsigned int* compressed_cpu = (unsigned int*)malloc(sizeof(int)* this->);
		// // copy compressed data
		// if(hipSuccess != hipMemcpy((void*)compressed_cpu, (void*)compressed_gpu, outputSize * sizeof(int), hipMemcpyDeviceToHost)){
		// 	std::cout << "Could not copy final output" << std::endl;
		// }

		// free gpu memory
		// hipFree((void*)compressed_gpu);
		// hipFree((void*)blockCounts_gpu);
		// hipFree((void*)orderArray_gpu);

		// get transfer time
		// hipEventCreate(&stop);
		// hipEventRecord(stop,0);
		// hipEventSynchronize(stop);
		// hipEventElapsedTime(&transferFromDeviceTime, start,stop);

		// write results to pointers if specified
	// 	if(pCompressionTime != NULL) (*pCompressionTime) = compressionTime;
	// 	if(pTransferToDeviceTime != NULL) (*pTransferToDeviceTime) = transferToDeviceTime;
	// 	if(ptranserFromDeviceTime != NULL) (*ptranserFromDeviceTime) = transferFromDeviceTime;
	// return compressed_cpu;
// }

//unsigned int* NoSortPackage::decompressData(){
//	// times to be measured
//		float transferToDeviceTime;
//		float compressionTime;
//		float transferFromDeviceTime;

		// start measuring time
		// CREATE_TIMER
		// START_TIMER

		// unsigned int *data_gpu, *result_gpu, *finalOutput_gpu, *output_cpu;
		// unsigned long long int* counts_gpu;
		// unsigned long long int blockCount = dataSize / 1024;

		// if(dataSize % 1024 > 0){
		// 	blockCount++;
		// }
		// hipMalloc((void**)&data_gpu, sizeof(int)*dataSize);
		// hipMalloc((void**)&counts_gpu, sizeof(unsigned long long int)*dataSize);
		// hipMemcpy(data_gpu, data, sizeof(int)*dataSize, hipMemcpyHostToDevice);

		// STOP_TIMER
		// GET_RESULT(transferToDeviceTime)
		// START_TIMER
	//	counts_cpu = (unsigned int*) malloc(sizeof(int)*dataSize);
//		dim3 blockDim(32, 32);
//		// get blocked sizes
//		getCounts<<<blockCount,blockDim>>>(data_gpu, counts_gpu, dataSize);
//		unsigned long long int lastBlockSize;
//		hipMemcpy(&lastBlockSize, counts_gpu  + (dataSize - 1), sizeof(unsigned long long int), hipMemcpyDeviceToHost);
//		// scan block sizes
//		thrust::device_ptr<unsigned long long int> countsPtr(counts_gpu);
//		// get counts
//		thrust::exclusive_scan(countsPtr, countsPtr + dataSize, countsPtr);
//		unsigned long long int lastOffset;
//	//	thrust::inclusive_scan(counts_cpu, counts_cpu + dataSize, counts_cpu);
//		hipMemcpy(&lastOffset, counts_gpu + (dataSize - 1), sizeof(unsigned long long int), hipMemcpyDeviceToHost);
//		unsigned long long int outputSize = lastBlockSize + lastOffset;
//		unsigned long long int realSize = 31*outputSize;
//
//		if(realSize % 32 > 0){
//			realSize /=32;
//			realSize++;
//		}
//		else{
//			realSize /=32;
//		}
//		SAFE_ASSIGN(outSize, realSize);
//	//	free(counts_cpu);
//		hipMalloc((void**)&result_gpu, sizeof(int) * outputSize);
//
//		decompressWords<<<blockCount,blockDim>>>(data_gpu, counts_gpu, result_gpu, dataSize);
//		hipFree(data_gpu);
//		hipFree(counts_gpu);
//
//		blockCount = outputSize / 1024;
//		if(dataSize % 1024 > 0){
//			blockCount++;
//		}
//
//		hipMalloc((void**)&finalOutput_gpu, sizeof(int)*outputSize);
//		mergeWords<<<blockCount,blockDim>>>(result_gpu, finalOutput_gpu, outputSize);
//		hipFree(result_gpu);

		// STOP_TIMER
		// GET_RESULT(compressionTime)
		// START_TIMER

		// output_cpu = (unsigned int*)malloc(sizeof(int) * outputSize);
		// hipMemcpy(output_cpu, finalOutput_gpu, sizeof(int) * outputSize, hipMemcpyDeviceToHost);
		// hipFree(finalOutput_gpu);
		// STOP_TIMER
		// GET_RESULT(transferFromDeviceTime)

		// SAFE_ASSIGN(pCompressionTime, compressionTime);
		// SAFE_ASSIGN(pTransferToDeviceTime, transferToDeviceTime);
		// SAFE_ASSIGN(ptranserFromDeviceTime, transferFromDeviceTime);

//	return output_cpu;
//}

template class NoSortPackage<unsigned long long int>;
template class NoSortPackage<unsigned int>;
