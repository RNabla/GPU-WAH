
#include <hip/hip_runtime.h>
///*
// * NoSortPackage.cpp
// *
// *  Created on: Feb 13, 2019
// *      Author: holgus103
// */
//
//#include "NoSortPackage.h"
//#include "compress.h"
//#include "kernels.h"
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//#include <math.h>
//#include <time.h>
//#include <stdlib.h>
//#include <thrust/remove.h>
//#include <thrust/device_ptr.h>
//#include "timeMeasuring.h"
//
//NoSortPackage::NoSortPackage() {
//
//}
//
//NoSortPackage::~NoSortPackage() {
//
//}
//
//void  NoSortPackage::compressData(unsigned int* data, unsigned long long int size){
//	// times to be measured
//		float transferToDeviceTime;
//		float compressionTime;
//		float transferFromDeviceTime;
//
//		// start measuring time
//		cudaEvent_t start, stop;
//		cudaEventCreate(&start);
//		cudaEventRecord(start,0);
//
//		unsigned long long blockCount = dataSize / (31*32);
//
//		if(dataSize % (31*32)> 0){
//			blockCount++;
//		}
//		// assign blockCount
//		this->orderingLength = blockCount;
//
//		unsigned int *data_gpu, *compressed_gpu;
//		unsigned long long int *blockCounts_gpu, *sizeCounter_gpu, *orderArray_gpu;
//
//		// calculate max output size (one extra bit for every 31 bits)
//		unsigned long long int maxExpectedSize = 8*sizeof(int)*dataSize;
//		if(maxExpectedSize % 31 > 0){
//			maxExpectedSize /= 31;
//			maxExpectedSize++;
//		}
//		else{
//			maxExpectedSize /= 31;
//		}
//
//		dim3 blockSize = dim3(32, 32, 1);
//
//		// allocate memory on the device
//		if(cudaSuccess != cudaMalloc((void**)&sizeCounter_gpu, sizeof(unsigned long long int))){
//			std::cout << "Could not allocate space for size counter" << std::endl;
//			FREE_ALL
//			return NULL;
//		}
//		if(cudaSuccess != cudaMalloc((void**)&orderArray_gpu, blockCount * sizeof(unsigned long long int))){
//			std::cout << "Could not allocate space for order array" << std::endl;
//			FREE_ALL
//			return NULL;
//		}
//		if(cudaSuccess != cudaMalloc((void**)&data_gpu, dataSize * sizeof(int))){
//			std::cout << "Could not allocate space for the data" << std::endl;
//			FREE_ALL
//			return NULL;
//		}
//		if(cudaSuccess != cudaMalloc((void**)&compressed_gpu, maxExpectedSize * sizeof(int))){
//			std::cout << "Could not allocate space for the compressed output" << std::endl;
//			FREE_ALL
//			return NULL;
//		}
//		if(cudaSuccess != cudaMalloc((void**)&blockCounts_gpu, blockCount* sizeof(unsigned long long int))){
//			std::cout << "Could not allocate space for the block sizes" << std::endl;
//			FREE_ALL
//			return NULL;
//		}
//
//		// copy input
//		if(cudaSuccess != cudaMemcpy(data_gpu, data_cpu, dataSize*sizeof(int), cudaMemcpyHostToDevice)){
//			std::cout << "Could not copy input" << std::endl;
//			FREE_ALL
//			return NULL;
//		}
//
//		// get transfer time
//		cudaEventCreate(&stop);
//		cudaEventRecord(stop,0);
//		cudaEventSynchronize(stop);
//		cudaEventElapsedTime(&transferToDeviceTime, start,stop);
//
//		// restart time measuring
//		cudaEventCreate(&start);
//		cudaEventRecord(start,0);
//
//		// call compression kernel
//		compressData<<<blockCount,blockSize>>>(data_gpu, compressed_gpu, blockCounts_gpu, orderArray_gpu, sizeCounter_gpu, dataSize);
//
//		// remove unnecessary data
//		cudaFree((void*)data_gpu);
//
//		// allocate memory for block sizes
//		this->blockSizes = (unsigned long long int*)malloc(sizeof(unsigned long long int) *blockCount);
//
//		// copy block sizes
//		if(cudaSuccess != cudaMemcpy(this->blockSizes, blockCounts_gpu, blockCount * sizeof(unsigned long long int), cudaMemcpyDeviceToHost)){
//			std::cout << "Could not copy last block counts" << std::endl;
//			FREE_ALL
//			return NULL;
//		}
//
//		// allocate ordering array
//		this->orderingArray = (unsigned long long int*) malloc(sizeof(unsigned long long int) * blockCount);
//
//		// copy ordering array
//		if(cudaSuccess != cudaMemcpy(this->orderingArray, orderArray_gpu, blockCount * sizeof(unsigned long long int), cudaMemcpyDeviceToHost)){
//			std::cout << "Could not copy ordering array" << std::endl;
//			FREE_ALL
//			return NULL;
//		}
//
//		unsigned long long int outputSize = 0;;
//
//		if(cudaSuccess != cudaMemcpy(&outputSize, sizeCounter_gpu, sizeof(unsigned long long int), cudaMemcpyDeviceToHost)){
//			std::cout << "Could not copy last block offset" << std::endl;
//			FREE_ALL
//			return NULL;
//		}
//
//		SAFE_ASSIGN(outSize, outputSize)
//
//		// get compression time
//		cudaEventCreate(&stop);
//		cudaEventRecord(stop,0);
//		cudaEventSynchronize(stop);
//		cudaEventElapsedTime(&compressionTime, start,stop);
//
//		// restart time measuring
//		cudaEventCreate(&start);
//		cudaEventRecord(start,0);
//
//		// allocate memory for results
//		unsigned int* compressed_cpu = (unsigned int*)malloc(sizeof(int)* outputSize);
//		// copy compressed data
//		if(cudaSuccess != cudaMemcpy((void*)compressed_cpu, (void*)compressed_gpu, outputSize * sizeof(int), cudaMemcpyDeviceToHost)){
//			std::cout << "Could not copy final output" << std::endl;
//		}
//
//		// free gpu memory
//		cudaFree((void*)compressed_gpu);
//		cudaFree((void*)blockCounts_gpu);
//		cudaFree((void*)orderArray_gpu);
//
//		// get transfer time
//		cudaEventCreate(&stop);
//		cudaEventRecord(stop,0);
//		cudaEventSynchronize(stop);
//		cudaEventElapsedTime(&transferFromDeviceTime, start,stop);
//
//		// write results to pointers if specified
//		if(pCompressionTime != NULL) (*pCompressionTime) = compressionTime;
//		if(pTransferToDeviceTime != NULL) (*pTransferToDeviceTime) = transferToDeviceTime;
//		if(ptranserFromDeviceTime != NULL) (*ptranserFromDeviceTime) = transferFromDeviceTime;
//	return compressed_cpu;
//}
//
//unsigned int* NoSortPackage::decompressData(){
//	// times to be measured
//		float transferToDeviceTime;
//		float compressionTime;
//		float transferFromDeviceTime;
//
//		// start measuring time
//		CREATE_TIMER
//		START_TIMER
//
//		unsigned int *data_gpu, *result_gpu, *finalOutput_gpu, *output_cpu;
//		unsigned long long int* counts_gpu;
//		unsigned long long int blockCount = dataSize / 1024;
//
//		if(dataSize % 1024 > 0){
//			blockCount++;
//		}
//		cudaMalloc((void**)&data_gpu, sizeof(int)*dataSize);
//		cudaMalloc((void**)&counts_gpu, sizeof(unsigned long long int)*dataSize);
//		cudaMemcpy(data_gpu, data, sizeof(int)*dataSize, cudaMemcpyHostToDevice);
//
//		STOP_TIMER
//		GET_RESULT(transferToDeviceTime)
//		START_TIMER
//	//	counts_cpu = (unsigned int*) malloc(sizeof(int)*dataSize);
//		dim3 blockDim(32, 32);
//		// get blocked sizes
//		getCounts<<<blockCount,blockDim>>>(data_gpu, counts_gpu, dataSize);
//		unsigned long long int lastBlockSize;
//		cudaMemcpy(&lastBlockSize, counts_gpu  + (dataSize - 1), sizeof(unsigned long long int), cudaMemcpyDeviceToHost);
//		// scan block sizes
//		thrust::device_ptr<unsigned long long int> countsPtr(counts_gpu);
//		// get counts
//		thrust::exclusive_scan(countsPtr, countsPtr + dataSize, countsPtr);
//		unsigned long long int lastOffset;
//	//	thrust::inclusive_scan(counts_cpu, counts_cpu + dataSize, counts_cpu);
//		cudaMemcpy(&lastOffset, counts_gpu + (dataSize - 1), sizeof(unsigned long long int), cudaMemcpyDeviceToHost);
//		unsigned long long int outputSize = lastBlockSize + lastOffset;
//		unsigned long long int realSize = 31*outputSize;
//
//		if(realSize % 32 > 0){
//			realSize /=32;
//			realSize++;
//		}
//		else{
//			realSize /=32;
//		}
//		SAFE_ASSIGN(outSize, realSize);
//	//	free(counts_cpu);
//		cudaMalloc((void**)&result_gpu, sizeof(int) * outputSize);
//
//		decompressWords<<<blockCount,blockDim>>>(data_gpu, counts_gpu, result_gpu, dataSize);
//		cudaFree(data_gpu);
//		cudaFree(counts_gpu);
//
//		blockCount = outputSize / 1024;
//		if(dataSize % 1024 > 0){
//			blockCount++;
//		}
//
//		cudaMalloc((void**)&finalOutput_gpu, sizeof(int)*outputSize);
//		mergeWords<<<blockCount,blockDim>>>(result_gpu, finalOutput_gpu, outputSize);
//		cudaFree(result_gpu);
//
//		STOP_TIMER
//		GET_RESULT(compressionTime)
//		START_TIMER
//
//		output_cpu = (unsigned int*)malloc(sizeof(int) * outputSize);
//		cudaMemcpy(output_cpu, finalOutput_gpu, sizeof(int) * outputSize, cudaMemcpyDeviceToHost);
//		cudaFree(finalOutput_gpu);
//		STOP_TIMER
//		GET_RESULT(transferFromDeviceTime)
//
//		SAFE_ASSIGN(pCompressionTime, compressionTime);
//		SAFE_ASSIGN(pTransferToDeviceTime, transferToDeviceTime);
//		SAFE_ASSIGN(ptranserFromDeviceTime, transferFromDeviceTime);
//
//	return output_cpu;
//}
