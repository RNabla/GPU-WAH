#include "hip/hip_runtime.h"
/*
 * compress.cu
 *
 *  Created on: 6 mai 2018
 *      Author: holgus103
 */
#include "compress.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <time.h>
#include <stdlib.h>  
#include <thrust/remove.h>
#include <thrust/device_ptr.h>


struct is_zero
{
	 __host__ __device__
	 bool operator()(const int x)
	 {
	   return x == 0;
	 }
};


// datasize is in integers!
unsigned int* compress(
		unsigned int* data_cpu,
		unsigned int dataSize,
		float* pTransferToDeviceTime,
		float* pCompressionTime,
		float* ptranserFromDeviceTime){

	// times to be measured
	float transferToDeviceTime;
	float compressionTime;
	float transferFromDeviceTime;

	// start measuring time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventRecord(start,0);

	int blockCount = dataSize / 1024;

	if(dataSize % 1024 > 0){
		blockCount++;
	}

	unsigned int *data_gpu, *compressed_gpu, *blockCounts_gpu, *finalOutput_gpu;

	// calculate max output size (one extra bit for every 31 bits)
	long long maxExpectedSize = 8*sizeof(int)*dataSize;
	if(maxExpectedSize % 31 > 0){
		maxExpectedSize /= 31;
		maxExpectedSize++;
	}
	else{
		maxExpectedSize /= 31;
	}

	maxExpectedSize *= 8*sizeof(int);

	dim3 blockSize = dim3(32, 32, 1);

	// allocate memory on the device
	hipMalloc((void**)&data_gpu, dataSize * sizeof(int));
	hipMalloc((void**)&compressed_gpu, maxExpectedSize * sizeof(int));
	hipMalloc((void**)&blockCounts_gpu, blockCount* sizeof(int));

	// copy input
	hipMemcpy(data_gpu, data_cpu, dataSize*sizeof(int), hipMemcpyHostToDevice);

	// get transfer time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&transferToDeviceTime, start,stop);

	// restart time measuring
	hipEventCreate(&start);
	hipEventRecord(start,0);

	// call compression kernel
	compressData<<<blockCount,blockSize>>>(data_gpu, compressed_gpu, blockCounts_gpu, dataSize);

	// remove unnecessary data
	hipFree((void*)data_gpu);
	thrust::device_ptr<unsigned int> blockCountsPtr(blockCounts_gpu);


	unsigned int lastWordNumber;
	hipMemcpy(&lastWordNumber, blockCounts_gpu + (blockCount - 1), sizeof(int), hipMemcpyDeviceToHost);
	thrust::exclusive_scan(blockCountsPtr, blockCountsPtr + blockCount, blockCountsPtr);
	unsigned int lastBlockOffset;
	hipMemcpy(&lastBlockOffset, blockCounts_gpu + (blockCount - 1), sizeof(int), hipMemcpyDeviceToHost);

	int outputSize = lastBlockOffset + lastWordNumber;
	hipMalloc((void**)&finalOutput_gpu, sizeof(int) * outputSize);
	// call merge kernel
	moveData<<<blockCount, blockSize>>>(compressed_gpu, finalOutput_gpu, blockCounts_gpu);

	// get compression time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&compressionTime, start,stop);

	// restart time measuring
	hipEventCreate(&start);
	hipEventRecord(start,0);

	// allocate memory for results
	unsigned int* compressed_cpu = (unsigned int*)malloc(sizeof(int)* outputSize);
	// copy compressed data
	hipMemcpy((void*)compressed_cpu, (void*)finalOutput_gpu, outputSize * sizeof(int), hipMemcpyDeviceToHost);

	// free gpu memory
	hipFree((void*)compressed_gpu);
	hipFree((void*)blockCounts_gpu);
	hipFree((void*)finalOutput_gpu);

	// get transfer time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&transferFromDeviceTime, start,stop);

	// write results to pointers if specified
	if(pCompressionTime != NULL) (*pCompressionTime) = compressionTime;
	if(pTransferToDeviceTime != NULL) (*pTransferToDeviceTime) = transferToDeviceTime;
	if(ptranserFromDeviceTime != NULL) (*ptranserFromDeviceTime) = transferFromDeviceTime;
	return compressed_cpu;
}



