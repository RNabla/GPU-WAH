#include "hip/hip_runtime.h"
/*
 * compress.cu
 *
 *  Created on: 6 mai 2018
 *      Author: holgus103
 */
#include "compress.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdlib.h>  
#include <thrust/remove.h>


struct is_zero
{
	 __host__ __device__
	 bool operator()(const int x)
	 {
	   return x == 0;
	 }
};


// datasize is in bytes!
unsigned int* compress(unsigned int* data_cpu, unsigned int dataSize){
	int blockCount = dataSize / (1024 *sizeof(int));

	if(dataSize % (1024*sizeof(int)) > 0){
		blockCount++;
	}

	unsigned int *data_gpu, *compressed_gpu;

	// calculate max output size (one extra bit for every 31 bits)
	long long maxExpectedSize = 8*sizeof(int)*dataSize;
	if(maxExpectedSize % 31 > 0){
		maxExpectedSize /= 31;
		maxExpectedSize++;
	}
	else{
		maxExpectedSize /= 31;
	}

	maxExpectedSize *= 8*sizeof(int);
	if(maxExpectedSize)

	dim3 dimBlock(32, 32);
	// allocate memory for results
	unsigned int* compressed_cpu = (unsigned int*)malloc(sizeof(int)*maxExpectedSize);
	// allocate memory on the device
	hipMalloc((void**)&data_gpu, dataSize * sizeof(int));
	hipMalloc((void**)&compressed_gpu, maxExpectedSize * sizeof(int));

	// copy input
	hipMemcpy(data_gpu, data_cpu, dataSize*sizeof(int), hipMemcpyHostToDevice);

	// call compression kernel
	compressData<<<blockCount,dimBlock>>>(data_gpu, compressed_gpu, dataSize);
//	compressed_gpu = thrust::remove_if(compressed_gpu, compressed_gpu + maxExpectedSize, is_zero());

	// copy compressed data
	hipMemcpy((void*)compressed_cpu, (void*)compressed_gpu, maxExpectedSize * sizeof(int), hipMemcpyDeviceToHost);
	
	// free gpu memory
	hipFree((void*)data_gpu);
	hipFree((void*)compressed_gpu);

	return compressed_cpu;
}



