#include "hip/hip_runtime.h"
/*
 * compress.cu
 *
 *  Created on: 6 mai 2018
 *      Author: holgus103
 */
#include "compress.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdlib.h>  
#include <thrust/remove.h>
#include <thrust/device_ptr.h>


struct is_zero
{
	 __host__ __device__
	 bool operator()(const int x)
	 {
	   return x == 0;
	 }
};


// datasize is in integers!
unsigned int* compress(unsigned int* data_cpu, unsigned int dataSize){
	int blockCount = dataSize / 1024;

	if(dataSize % 1024 > 0){
		blockCount++;
	}

	unsigned int *data_gpu, *compressed_gpu, *blockCounts_gpu, *finalOutput_gpu;

	// calculate max output size (one extra bit for every 31 bits)
	long long maxExpectedSize = 8*sizeof(int)*dataSize;
	if(maxExpectedSize % 31 > 0){
		maxExpectedSize /= 31;
		maxExpectedSize++;
	}
	else{
		maxExpectedSize /= 31;
	}

	maxExpectedSize *= 8*sizeof(int);

	dim3 blockSize = dim3(32, 32, 1);
	// allocate memory on the device
	hipMalloc((void**)&data_gpu, dataSize * sizeof(int));
	hipMalloc((void**)&compressed_gpu, maxExpectedSize * sizeof(int));
	hipMalloc((void**)&blockCounts_gpu, blockCount* sizeof(int));
	// copy input
	hipMemcpy(data_gpu, data_cpu, dataSize*sizeof(int), hipMemcpyHostToDevice);

	// call compression kernel
	compressData<<<blockCount,blockSize>>>(data_gpu, compressed_gpu, blockCounts_gpu, dataSize);
	// remove unnecessary data
	hipFree((void*)data_gpu);
	thrust::device_ptr<unsigned int> blockCountsPtr(blockCounts_gpu);
	unsigned int* wordNumbers = (unsigned int*)malloc(sizeof(int)*blockCount);
	hipMemcpy(wordNumbers, blockCounts_gpu, sizeof(int) *blockCount, hipMemcpyDeviceToHost);
	thrust::exclusive_scan(blockCountsPtr, blockCountsPtr + blockCount, blockCountsPtr);
	thrust::inclusive_scan(wordNumbers, wordNumbers + blockCount, wordNumbers);
	hipMalloc((void**)&finalOutput_gpu, sizeof(int) * wordNumbers[blockCount-1]);
	// call merge kernel
	moveData<<<blockCount, blockSize>>>(compressed_gpu, finalOutput_gpu, blockCounts_gpu);
	// allocate memory for results
	unsigned int* compressed_cpu = (unsigned int*)malloc(sizeof(int)*wordNumbers[blockCount-1]);
	// copy compressed data
	hipMemcpy((void*)compressed_cpu, (void*)finalOutput_gpu, wordNumbers[blockCount-1] * sizeof(int), hipMemcpyDeviceToHost);

	// free gpu memory

	hipFree((void*)compressed_gpu);
	hipFree((void*)blockCounts_gpu);
	hipFree((void*)finalOutput_gpu);
	free(wordNumbers);
	return compressed_cpu;
}



