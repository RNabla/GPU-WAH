#include "hip/hip_runtime.h"
/*
 * compress.cu
 *
 *  Created on: 6 mai 2018
 *      Author: holgus103
 */
#include "compress.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdlib.h>  

int* compress(int* data_cpu, int dataSize){
	int* data_gpu,* compressed_gpu, int* warpInfo_gpu;

	// calculate max output size (one extra bit for every 31 bits)
	long long maxExpectedSize = 8*sizeof(int)*dataSize;
	maxExpectedSize *= 32;
	maxExpectedSize /= 31;
	maxExpectedSize /= 8*sizeof(int);

	// increment in case it got rounded
	maxExpectedSize++;

	// allocate memory for results
	int* compressed_cpu = (int*)malloc(sizeof(int)*maxExpectedSize);

	// allocate memory on the device
	hipMalloc((void**)&data_gpu, dataSize * sizeof(int));
	hipMalloc((void**)&compressed_gpu, maxExpectedSize * sizeof(int));
	hipMalloc((void**)&warpInfo_gpu, 1 * sizeof(int));

	// copy input
	hipMemcpy(data_gpu, data_cpu, dataSize*sizeof(int), hipMemcpyHostToDevice);

	// call compression kernel
	compressData<<<1,32>>>(data_gpu, compressed_gpu, warpInfo_gpu);

	// copy compressed data
	hipMemcpy((void*)compressed_cpu, (void*)compressed_gpu, maxExpectedSize * sizeof(int), hipMemcpyDeviceToHost);
	
	// free gpu memory
	hipFree((void*)data_gpu);
	hipFree((void*)compressed_gpu);
	hipFree((void*)warpInfo_gpu);

	return compressed_cpu;
}



