#include "hip/hip_runtime.h"
/*
 * compress.cu
 *
 *  Created on: 6 mai 2018
 *      Author: holgus103
 */
#include "compress.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <time.h>
#include <stdlib.h>  
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include "timeMeasuring.h"

#define FREE_ALL \
	if(data_gpu) hipFree(data_gpu);\
	if(compressed_gpu) hipFree(compressed_gpu);\
	if(blockCounts_gpu) hipFree(blockCounts_gpu);\
	if(finalOutput_gpu) hipFree(finalOutput_gpu);\
	if(orderArray_gpu) hipFree(orderArray_gpu);\
	if(sizeCounter_gpu) hipFree(sizeCounter_gpu);


struct is_zero
{
	 __host__ __device__
	 bool operator()(const int x)
	 {
	   return x == 0;
	 }
};



// datasize is in integers!
unsigned int* compress(
		unsigned int* data_cpu,
		unsigned int dataSize,
		unsigned int* outSize,
		unsigned int** orderingArray,
		float* pTransferToDeviceTime,
		float* pCompressionTime,
		float* ptranserFromDeviceTime){

	// times to be measured
	float transferToDeviceTime;
	float compressionTime;
	float transferFromDeviceTime;

	// start measuring time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventRecord(start,0);

	int blockCount = dataSize / (31*32);

	if(dataSize % (31*32)> 0){
		blockCount++;
	}

	unsigned int *data_gpu, *compressed_gpu, *blockCounts_gpu, *finalOutput_gpu, *orderArray_gpu, *sizeCounter_gpu;

	// calculate max output size (one extra bit for every 31 bits)
	long long maxExpectedSize = 8*sizeof(int)*dataSize;
	if(maxExpectedSize % 31 > 0){
		maxExpectedSize /= 31;
		maxExpectedSize++;
	}
	else{
		maxExpectedSize /= 31;
	}

	dim3 blockSize = dim3(32, 32, 1);

	// allocate memory on the device
	if(hipSuccess != hipMalloc((void**)&sizeCounter_gpu, sizeof(int))){
		std::cout << "Could not allocate space for size counter" << std::endl;
		FREE_ALL
		return NULL;
	}
	if(hipSuccess != hipMalloc((void**)&orderArray_gpu, blockCount * sizeof(int))){
		std::cout << "Could not allocate space for order array" << std::endl;
		FREE_ALL
		return NULL;
	}
	if(hipSuccess != hipMalloc((void**)&data_gpu, dataSize * sizeof(int))){
		std::cout << "Could not allocate space for the data" << std::endl;
		FREE_ALL
		return NULL;
	}
	if(hipSuccess != hipMalloc((void**)&compressed_gpu, maxExpectedSize * sizeof(int))){
		std::cout << "Could not allocate space for the compressed output" << std::endl;
		FREE_ALL
		return NULL;
	}
	if(hipSuccess != hipMalloc((void**)&blockCounts_gpu, blockCount* sizeof(int))){
		std::cout << "Could not allocate space for the block sizes" << std::endl;
		FREE_ALL
		return NULL;
	}

	// copy input
	if(hipSuccess != hipMemcpy(data_gpu, data_cpu, dataSize*sizeof(int), hipMemcpyHostToDevice)){
		std::cout << "Could not copy input" << std::endl;
		FREE_ALL
		return NULL;
	}

	// get transfer time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&transferToDeviceTime, start,stop);

	// restart time measuring
	hipEventCreate(&start);
	hipEventRecord(start,0);

	// call compression kernel
	compressData<<<blockCount,blockSize>>>(data_gpu, compressed_gpu, blockCounts_gpu, orderArray_gpu, sizeCounter_gpu, dataSize);

	// remove unnecessary data
	hipFree((void*)data_gpu);


	unsigned int* orderArray = (unsigned int*)malloc(sizeof(int) * blockCount);
	(*orderingArray) = orderArray;
	if(hipSuccess != hipMemcpy(&orderArray, orderArray_gpu, blockCount * sizeof(int), hipMemcpyDeviceToHost)){
		std::cout << "Could not copy last block count" << std::endl;
		FREE_ALL
		return NULL;
	}

	unsigned int outputSize = 0;;

	if(hipSuccess != hipMemcpy(&outputSize, sizeCounter_gpu, sizeof(int), hipMemcpyDeviceToHost)){
		std::cout << "Could not copy last block offset" << std::endl;
		FREE_ALL
		return NULL;
	}

	SAFE_ASSIGN(outSize, outputSize)
	if(hipSuccess != hipMalloc((void**)&finalOutput_gpu, sizeof(int) * outputSize)){
		std::cout << "Could not allocate final Output" << std::endl;
		FREE_ALL
		return NULL;
	}

	// get compression time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&compressionTime, start,stop);

	// restart time measuring
	hipEventCreate(&start);
	hipEventRecord(start,0);

	// allocate memory for results
	unsigned int* compressed_cpu = (unsigned int*)malloc(sizeof(int)* outputSize);
	// copy compressed data
	if(hipSuccess != hipMemcpy((void*)compressed_cpu, (void*)finalOutput_gpu, outputSize * sizeof(int), hipMemcpyDeviceToHost)){
		std::cout << "Could not copy final output" << std::endl;
	}

	// free gpu memory
	hipFree((void*)compressed_gpu);
	hipFree((void*)blockCounts_gpu);
	hipFree((void*)finalOutput_gpu);
	hipFree((void*)orderArray_gpu);

	// get transfer time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&transferFromDeviceTime, start,stop);

	// write results to pointers if specified
	if(pCompressionTime != NULL) (*pCompressionTime) = compressionTime;
	if(pTransferToDeviceTime != NULL) (*pTransferToDeviceTime) = transferToDeviceTime;
	if(ptranserFromDeviceTime != NULL) (*ptranserFromDeviceTime) = transferFromDeviceTime;
	return compressed_cpu;
}



