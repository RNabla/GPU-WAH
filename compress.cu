#include "hip/hip_runtime.h"
/*
 * compress.cu
 *
 *  Created on: 6 mai 2018
 *      Author: holgus103
 */
#include "compress.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <time.h>
#include <stdlib.h>  
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include "timeMeasuring.h"


struct is_zero
{
	 __host__ __device__
	 bool operator()(const int x)
	 {
	   return x == 0;
	 }
};


// datasize is in integers!
unsigned int* compress(
		unsigned int* data_cpu,
		unsigned long long int dataSize,
		unsigned long long int* outSize,
		float* pTransferToDeviceTime,
		float* pCompressionTime,
		float* ptranserFromDeviceTime){

	// times to be measured
	float transferToDeviceTime;
	float compressionTime;
	float transferFromDeviceTime;

	// start measuring time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventRecord(start,0);

	int blockCount = dataSize / (31*32);

	if(dataSize % (31*32)> 0){
		blockCount++;
	}

	unsigned int *data_gpu, *compressed_gpu, *finalOutput_gpu;
	unsigned long long int* blockCounts_gpu;

	// calculate max output size (one extra bit for every 31 bits)
	unsigned long long int maxExpectedSize = 8*sizeof(int)*dataSize;
	if(maxExpectedSize % 31 > 0){
		maxExpectedSize /= 31;
		maxExpectedSize++;
	}
	else{
		maxExpectedSize /= 31;
	}

	dim3 blockSize = dim3(32, 32, 1);

	// allocate memory on the device
	if(hipSuccess != hipMalloc((void**)&data_gpu, dataSize * sizeof(int))){
		std::cout << "Could not allocate space for the data" << std::endl;
		return NULL;
	}
	if(hipSuccess != hipMalloc((void**)&compressed_gpu, maxExpectedSize * sizeof(int))){
		std::cout << "Could not allocate space for the compressed output" << std::endl;
		hipFree(data_gpu);
		return NULL;
	}
	if(hipSuccess != hipMalloc((void**)&blockCounts_gpu, blockCount* sizeof(unsigned long long int))){
		std::cout << "Could not allocate space for the block sizes" << std::endl;
		hipFree(data_gpu);
		hipFree(compressed_gpu);
		return NULL;
	}

	// copy input
	if(hipSuccess != hipMemcpy(data_gpu, data_cpu, dataSize*sizeof(int), hipMemcpyHostToDevice)){
		std::cout << "Could not copy input" << std::endl;
		hipFree(data_gpu);
		hipFree(compressed_gpu);
		hipFree(blockCounts_gpu);
		return NULL;
	}

	// get transfer time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&transferToDeviceTime, start,stop);

	// restart time measuring
	hipEventCreate(&start);
	hipEventRecord(start,0);

	// call compression kernel
	compressData<<<blockCount,blockSize>>>(data_gpu, compressed_gpu, blockCounts_gpu, dataSize);

	// remove unnecessary data
	hipFree((void*)data_gpu);
	thrust::device_ptr<unsigned long long int> blockCountsPtr(blockCounts_gpu);


	unsigned long long int lastWordNumber;

	if(hipSuccess != hipMemcpy(&lastWordNumber, blockCounts_gpu + (blockCount - 1), sizeof(unsigned long long int), hipMemcpyDeviceToHost)){
		std::cout << "Could not copy last block count" << std::endl;
		hipFree(compressed_gpu);
		hipFree(blockCounts_gpu);
		return NULL;
	}

	thrust::exclusive_scan(blockCountsPtr, blockCountsPtr + blockCount, blockCountsPtr);
	unsigned long long int  lastBlockOffset;

	if(hipSuccess != hipMemcpy(&lastBlockOffset, blockCounts_gpu + (blockCount - 1), sizeof(unsigned long long int), hipMemcpyDeviceToHost)){
		std::cout << "Could not copy last block offset" << std::endl;
		hipFree(compressed_gpu);
		hipFree(blockCounts_gpu);
		return NULL;
	}

	unsigned long long int outputSize = lastBlockOffset + lastWordNumber;
	SAFE_ASSIGN(outSize, outputSize)
	if(hipSuccess != hipMalloc((void**)&finalOutput_gpu, sizeof(int) * outputSize)){
		std::cout << "Could not allocate final Output" << std::endl;
		hipFree(compressed_gpu);
		hipFree(blockCounts_gpu);
		return NULL;
	}
	// call merge kernel
	moveData<<<blockCount, blockSize>>>(compressed_gpu, finalOutput_gpu, blockCounts_gpu);

	// get compression time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&compressionTime, start,stop);

	// restart time measuring
	hipEventCreate(&start);
	hipEventRecord(start,0);

	// allocate memory for results
	unsigned int* compressed_cpu = (unsigned int*)malloc(sizeof(int)* outputSize);
	// copy compressed data
	if(hipSuccess != hipMemcpy((void*)compressed_cpu, (void*)finalOutput_gpu, outputSize * sizeof(int), hipMemcpyDeviceToHost)){
		std::cout << "Could not copy final output" << std::endl;
	}

	// free gpu memory
	hipFree((void*)compressed_gpu);
	hipFree((void*)blockCounts_gpu);
	hipFree((void*)finalOutput_gpu);

	// get transfer time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&transferFromDeviceTime, start,stop);

	// write results to pointers if specified
	if(pCompressionTime != NULL) (*pCompressionTime) = compressionTime;
	if(pTransferToDeviceTime != NULL) (*pTransferToDeviceTime) = transferToDeviceTime;
	if(ptranserFromDeviceTime != NULL) (*ptranserFromDeviceTime) = transferFromDeviceTime;
	return compressed_cpu;
}



