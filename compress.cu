#include "hip/hip_runtime.h"
/*
 * compress.cu
 *
 *  Created on: 6 mai 2018
 *      Author: holgus103
 */
#include "compress.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdlib.h>  

int* compress(int* data_cpu, int dataSize){
	int* data_gpu,* compressed_gpu;

	// allocate memory for results
	int* compressed_cpu = (int*)malloc(sizeof(int)*dataSize);

	// calculate max output size (one extra bit for every 31 bits)
	long long maxExpectedSize = 8*sizeof(int)*dataSize;
	maxExpectedSize *= 32;
	maxExpectedSize /= 31;
	maxExpectedSize /= 8*sizeof(int);

	// increment in case it got rounded
	maxExpectedSize++;

	// allocate memory on the device
	hipMalloc((void**)&data_gpu, dataSize * sizeof(int));
	hipMalloc((void**)&compressed_gpu, maxExpectedSize * sizeof(int));

	// copy input
	hipMemcpy(data_gpu, data_gpu, dataSize*sizeof(int), hipMemcpyHostToDevice);

	// call compression kernel
	compressData<<<1,1>>>(data_gpu, compressed_gpu);

	// copy compressed data
	hipMemcpy((void*)compressed_cpu, (void*)compressed_gpu, maxExpectedSize * sizeof(int), hipMemcpyDeviceToHost);
	
	// free gpu memory
	hipFree((void*)data_gpu);
	hipFree((void*)compressed_gpu);

	return compressed_cpu;
}



